#include "hip/hip_runtime.h"
#include <chrono>

#include "Tensor.h"
#include "cpu_kernels.h"
#include "cuda_kernels.h"
#include "common.h"


#define THREADS_PER_BLOCK 64

Tensor::Tensor(floatX* _data, int* _shape, int _ndim, std::string _device)
{
    //throw py::value_error("Creating a tensor");
    shape = _shape;
    ndim = _ndim;

    // Calculate total amount of elements in the tensor
    size = 1;
    for (int i = 0; i < ndim; i++)
    {
        size *= shape[i];
    }

    device = _device;
    data = _data;
    // If device change is needed
    if (_device == "cuda" && check_pointer_location(data) == "cpu")
    {
        data = data_to_cuda(data);
    }
    else if (_device == "cpu" && check_pointer_location(data) == "cuda")
    {
        data = data_to_cpu(data);
    }

    // Allocate memory for strides which has "ndim" elements
    strides = new int[ndim];

    // Calculate stride for each dimension
    int stride = 1;
    for (int i = ndim - 1; i >= 0; i--)
    {
        strides[i] = stride;
        stride *= shape[i];
    }
}

Tensor::~Tensor()
{
    if (device == "cuda")
    {
        cuda_check(hipFree(data));
    }
    else if (device == "cpu")
    {
        delete[] data;
    }

    delete[] strides;
    delete[] shape;
}

Tensor* Tensor::add_tensors(Tensor* _tensor1, Tensor* _tensor2)
{
    if (_tensor1->ndim != _tensor2->ndim)
    {
        throw_error("Tensors must have the same number of dimensions for addition. Current dimensions: %d and %d\n", _tensor1->ndim, _tensor2->ndim);
    }

    if (_tensor1->device != _tensor2->device)
    {
        throw_error("Tensors must be on the same device. Current devices: %s and %s\n", _tensor1->device.c_str(), _tensor2->device.c_str());
    }

    int ndim = _tensor1->ndim;
    std::string device = _tensor1->device;
    int* shape = new int[ndim];

    for (int i = 0; i < ndim; i++)
    {
        if (_tensor1->shape[i] != _tensor2->shape[i])
        {
            throw_error("Tensors must have the same shape for addition. Current shape at index %d: %d and %d\n", i, _tensor1->shape[i], _tensor2->shape[i]);
        }
        shape[i] = _tensor1->shape[i];
    }

    if (device == "cuda")
    {
        floatX* data;
        cuda_check(hipMalloc((void**)&data, _tensor1->size * sizeof(floatX)));

        int num_blocks = (_tensor1->size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        add_cuda << <num_blocks, THREADS_PER_BLOCK >> > (_tensor1->data, _tensor2->data, data, _tensor1->size);

        cuda_check(hipGetLastError());
        cuda_check(hipDeviceSynchronize());
        return new Tensor(data, shape, ndim, device);
    }
    else
    {
        floatX* data = new floatX[_tensor1->size];
        add_cpu(_tensor1->data, _tensor2->data, data, _tensor1->size);
        return new Tensor(data, shape, ndim, device);
    }
}

Tensor* Tensor::arange(int _start, int _end, std::string _device)
{
    int start = _start;
    int end = _end;
    int size = end - start;
    // Size and shape equal in 1d tensor
    int* shape = new int[1];
    shape[0] = size;

    std::string device = _device;

    if (_device == "cuda")
    {
        floatX* data;
        cuda_check(hipMalloc((void**)&data, size * sizeof(floatX)));

        int num_blocks = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        arange_cuda << <num_blocks, THREADS_PER_BLOCK >> > (data, start, size);
        cuda_check(hipGetLastError());
        return new Tensor(data, shape, 1, device);
    }
    else
    {
        floatX* data = new floatX[size];
        arange_cpu(data, start, size);
        return new Tensor(data, shape, 1, device);
    }
}

floatX Tensor::get_item(int* _indicies)
{
    // Convert n-dimensional indicies to 1 index to be used with a 1d array
    int index = 0;
    for (int i = 0; i < ndim; i++)
    {
        index += _indicies[i] * strides[i];
    }

    if ((index >= size) || (index < 0))
    {
        throw_error("Index should be less than the size of tensor and greater than 0, current index and size are: %d, %d\n", index, size);
    }

    floatX result;
    if (device == "cuda")
    {
        floatX* tmp_cpu_data = data_to_cpu(data, false);
        result = tmp_cpu_data[index];
        delete[] tmp_cpu_data;
    }
    else if (device == "cpu")
    {
        result = data[index];
    }

    return result;
}

Tensor* Tensor::to(std::string _device)
{
    if (device == "cpu" && _device == "cuda")
    {
        return new Tensor(data, shape, ndim, "cuda");
    }
    else if (device == "cuda" && _device == "cpu")
    {
        return new Tensor(data, shape, ndim, "cpu");
    }
    return this;
}

std::string Tensor::tensor_to_string()
{
    std::string result;
    result += "tensor(";
    // Set indent level to num of characters in "tensor("
    result += data_to_string(0, 0, 7);
    result += ", device='" + device + "'";
    result += ")";
    return result;
}

std::string Tensor::data_to_string(int _dim, int _offset, int _indentLevel)
{
    std::string result;
    if (_dim == ndim - 1) {
        // Print the innermost dimension (base case)
        result += "[";
        for (int i = 0; i < shape[_dim]; ++i) {
            if (device == "cuda")
            {
                __hip_bfloat16* cpu_data = new __hip_bfloat16;
                cuda_check(hipMemcpy(cpu_data, &data[_offset + i], sizeof(__hip_bfloat16), hipMemcpyDeviceToHost));

#if defined(ENABLE_FP32)
                result += std::to_string(*cpu_data);
#elif defined(ENABLE_FP16)
                result += std::to_string(__half2float(*cpu_data));
#else
                result += std::to_string(__bfloat162float(*cpu_data));
#endif
                
                delete cpu_data;
            }
            else
            {
#if defined(ENABLE_FP32)
                result += std::to_string(data[_offset + i]);
#elif defined(ENABLE_FP16)
                result += std::to_string(__half2float(data[_offset + i]));
#else
                result += std::to_string(__bfloat162float(data[_offset + i]));
#endif

                
            }
            if (i != shape[_dim] - 1) {
                result += ", ";
            }
        }
        result += "]";
    }
    else {
        // Recursively print each slice of the current dimension
        result += "[";
        size_t stride = 1;
        for (size_t i = _dim + 1; i < ndim; ++i) {
            stride *= shape[i];
        }
        for (int i = 0; i < shape[_dim]; ++i) {
            if (i != 0) {
                result += "\n" + std::string(_indentLevel + 1, ' ');
            }
            result += data_to_string(_dim + 1, _offset + i * stride, _indentLevel + 1);
            if (i != shape[_dim] - 1) {
                result += ",";
            }
        }
        result += "]";
    }
    return result;
}

floatX* Tensor::data_to_cuda(floatX* _data)
{
    floatX* cuda_data;
    cuda_check(hipMalloc((void**)&cuda_data, size * sizeof(floatX)));
    cuda_check(hipMemcpy(cuda_data, _data, size * sizeof(floatX), hipMemcpyHostToDevice));

    delete[] _data;

    return cuda_data;
}

floatX* Tensor::data_to_cpu(floatX* _data, bool _delete_original)
{
    floatX* cpu_data = new floatX[size];
    cuda_check(hipMemcpy(cpu_data, _data, size * sizeof(floatX), hipMemcpyDeviceToHost));
    if (_delete_original)
    {
        cuda_check(hipFree(_data));
    }

    return cpu_data;
}

std::string Tensor::check_pointer_location(void* _ptr)
{
    hipPointerAttribute_t attributes;
    hipError_t error = hipPointerGetAttributes(&attributes, _ptr);
    if (error == hipSuccess)
    {
        if (attributes.type == hipMemoryTypeDevice)
        {
            return "cuda";
        }
        else if (attributes.type == hipMemoryTypeHost || attributes.type == cudaMemoryTypeUnregistered)
        {
            return "cpu";
        }
        else
        {
            throw_error("Can't determine poiner location.");
        }
    }
    else
    {
        throw_error("Cuda error: %s\n", hipGetErrorString(error));
    }
}