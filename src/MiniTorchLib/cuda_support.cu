#include <hip/hip_runtime.h>
#include "cuda_support.h"

bool cuda_support::is_available()
{
	int deviceCount = 0;
	hipError_t error = hipGetDeviceCount(&deviceCount);

	return error == hipSuccess;
}