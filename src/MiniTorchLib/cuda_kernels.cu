#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_kernels.h"

__global__ void add_cuda(floatX* _data1, floatX* _data2, floatX* _result_data, int _size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < _size)
	{
		_result_data[i] = _data1[i] + _data2[i];
	}
}

__global__ void arange_cuda(floatX* _data, int _start, int _size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < _size)
	{
		_data[i] = i + _start;
	}
}